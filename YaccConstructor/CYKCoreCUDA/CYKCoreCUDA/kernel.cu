
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
using namespace std;

typedef unsigned long long uint64;
typedef unsigned long uint16;

typedef uint64 rule;

typedef uint16 cellData;

//(16|   16| 16|8       |       8  |)
//(A |->  B|  C|<lblName|lblWeight>|)
__device__ int getRuleA(rule r){return r >> 48;}
__device__ int getRuleB(rule r){return r >> 32 & 0xFFFF;}
__device__ int getRuleC(rule r){return r >> 16 & 0xFFFF;}
__device__ int getRuleN(rule r){return r >> 8 & 0xFF;}
__device__ int getRuleW(rule r){return r & 0xFF;}

__host__ rule buildRule(int A, int B, int C, int lblName, int lblWeght){
	return (uint64)A << 48 | (uint64)B << 32 | (uint64)C << 16 | lblName << 8 | lblWeght;
}

//(16 |16                |16       |8       |8        )
//(k  |non-terminalIndex |lblState |lblName |lblWeght )
__device__ int getDataI(cellData d){return d;}

__host__ cellData buildData_Host(int ruleIndex){return ruleIndex;}
__device__ cellData buildData(int k, int ruleIndex, int lblState, int lblName, int lblWeght){
	return ruleIndex;
}

__global__ void processRule(rule* rules, int rulesCount, int nCount, int strLen, int subLen, cellData* table){
	// subLen === l
	// start === i
	int start = blockIdx.y * blockDim.y + threadIdx.y;
	if(start >= strLen - subLen) return;
	rule currentRule = rules[threadIdx.x];

	for(int k = 0; k < subLen; k++){	
		cellData *current = table + (			subLen * strLen + start			  ) * (nCount + 1);
		cellData *left	  = table + (				 k * strLen + start			  ) * (nCount + 1);
		cellData *right	  = table + ( (subLen - k - 1) * strLen + (k + start + 1) ) * (nCount + 1);
		
		int c = getRuleC(currentRule);
		if(current[ getRuleA(currentRule) ]) return;

		for(int m = 1; m <= nCount; m++){
			if ( getDataI( left[m] ) == getRuleB(currentRule) ){
				for(int n = 1; n <= nCount; n++){
					if ( getDataI( right[n] ) == c ){							
						current[ getRuleA(currentRule) ] = getRuleA(currentRule);
					}
				}
			}	
		}
	}
	__syncthreads();
}

__host__ void fillTable(rule* rules, int rulesCount, int nCount, int strLen, cellData* table){
	hipEvent_t start, stop;
    float gpuTime = 0.0f;

	int deviceCount;
	hipDeviceProp_t cdp;
	hipGetDeviceProperties ( &cdp, 0 );

    hipEventCreate ( &start );
    hipEventCreate ( &stop );
	hipEventRecord ( start, 0 );

    cellData *dev_table = 0;
	rule *dev_rules = 0;
	int table_size = strLen * strLen * (nCount+1);
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);

    cudaStatus = hipMalloc((void**)&dev_table, table_size * sizeof(cellData));
    cudaStatus = hipMalloc((void**)&dev_rules, rulesCount * sizeof(rule));
    cudaStatus = hipMemcpy(dev_table, table, table_size * sizeof(cellData), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_rules, rules, rulesCount * sizeof(rule), hipMemcpyHostToDevice);

	int threadsPerBlockX = cdp.maxThreadsPerBlock / rulesCount;

	for(int subLen = 1; subLen <= strLen; subLen++){
		processRule<<< dim3( 1,(strLen-subLen)/(threadsPerBlockX)+1 ), dim3(rulesCount,  threadsPerBlockX ) >>>(dev_rules, rulesCount, nCount, strLen, subLen, dev_table);
	}

    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(table, dev_table, table_size * sizeof(cellData), hipMemcpyDeviceToHost);

	hipEventRecord (stop, 0);

	hipEventSynchronize ( stop );
	hipEventElapsedTime ( &gpuTime, start, stop );

	cout<<"time "<< gpuTime<<endl;
	
    hipFree(dev_table);
    hipFree(dev_rules);
}

int main(){
	int rulesCount = 7;
	int wordLen = 1000;
	int nCount = 4;

	rule *rules = new rule[rulesCount];
	rules[0] = buildRule(1,2,3,0,0);
	rules[1] = buildRule(2,3,2,0,0);
	rules[2] = buildRule(2,3,3,0,0);
	rules[3] = buildRule(3,1,2,0,0);
	rules[4] = buildRule(4,2,4,0,0);
	rules[5] = buildRule(1,4,2,0,0);
	rules[6] = buildRule(3,4,2,0,0);

	cellData* table = new cellData[wordLen * wordLen * (nCount + 1)];
	for(int i = 0; i < wordLen * wordLen * (nCount + 1); i++)
		table[i] = 0;

	for(int i=0; i<wordLen; i++){
		table[i*(nCount+1) + 0] = 1;
		table[i*(nCount+1) + 3] = buildData_Host(3);
	}

	table[(wordLen-2)*(nCount+1) + 0] = 1;
	table[(wordLen-2)*(nCount+1) + 2] = buildData_Host(2);
	table[(wordLen-2)*(nCount+1) + 3] = buildData_Host(0);

	table[(wordLen-5)*(nCount+1) + 0] = 1;
	table[(wordLen-5)*(nCount+1) + 4] = buildData_Host(4);
	table[(wordLen-5)*(nCount+1) + 3] = buildData_Host(0);

	fillTable(rules, rulesCount, nCount, wordLen, table);

	//for(int i = 0; i < wordLen; i++){
	//	for(int j = 0; j < wordLen*(nCount+1); j++){
	//		if( !( j % (nCount+1) ) && j ) cout<<' ';
	//		if( !( j % (nCount+1) ) )
	//			cout<<"";//(table[i*wordLen*(nCount+1)+j]);
	//		else
	//			cout<<(table[i*wordLen*(nCount+1)+j]);		
	//		//if( !( j % (nCount+1) ) ) cout<<' ';
	//	}
	//	cout<<endl;
	//}

	for(int i = wordLen-1; i < wordLen; i++){
		for(int j = 0; j < 1*(nCount+1); j++){
			if( !( j % (nCount+1) ) && j ) cout<<' ';
			if( !( j % (nCount+1) ) )
				cout<<(table[i*wordLen*(nCount+1)+j]);
			else
				cout<<(table[i*wordLen*(nCount+1)+j] >> 32 & 0xFFFF);		
			if( !( j % (nCount+1) ) ) cout<<' ';
		}
		cout<<endl;
	}

	system("pause");
	return 0;
}