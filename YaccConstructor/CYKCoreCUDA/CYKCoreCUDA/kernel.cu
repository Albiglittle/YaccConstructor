
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
using namespace std;

typedef unsigned long long uint64;
typedef unsigned long uint32;

typedef uint64 rule;
typedef uint32 token;

typedef uint64 cellData;

//(16|   16| 16|8       |       8  |)
//(A |->  B|  C|<lblName|lblWeight>|)
__device__ int getRuleA(rule r){return r >> 48;}
__device__ int getRuleB(rule r){return r >> 32 & 0xFFFF;}
__device__ int getRuleC(rule r){return r >> 16 & 0xFFFF;}
__device__ int getRuleN(rule r){return r >> 8 & 0xFF;}
__device__ int getRuleW(rule r){return r & 0xFF;}

__host__ cellData buildRule(int A, int B, int C, int lblName, int lblWeght){
	return (uint64)A << 48 | (uint64)B << 32 | (uint64)C << 16 | lblName << 8 | lblWeght;
}

//(16 |16                |16       |8       |8        )
//(k  |non-terminalIndex |lblState |lblName |lblWeght )
__device__ int getDataK(cellData d){return d >> 48;}
__device__ int getDataI(cellData d){return d >> 32 & 0xFFFF;}
__device__ int getDataS(cellData d){return d >> 16 & 0xFFFF;}
__device__ int getDataN(cellData d){return d >> 8 & 0xFF;}
__device__ int getDataW(cellData d){return d & 0xFF;}


__host__ cellData buildData_Host(int ruleIndex){return (uint64)ruleIndex << 32;}
__device__ cellData buildData(int k, int ruleIndex, int lblState, int lblName, int lblWeght){
	return (uint64)k << 48 | (uint64)ruleIndex << 32 | lblState << 16 | lblName << 8 | lblWeght;
}

__device__ int getCellRuleTop (rule* rules, cellData* cellContent, int i){
	return getDataI( cellContent[ cellContent[0] - i ] );
}

__device__ void processRule(rule* rules, int rulesCount, rule currentRule, int start, int k, int strLen, int subLen, cellData* table){
	// subLen <-- l
	// start <-- i

	int b = getRuleB(currentRule);
	int c = getRuleC(currentRule);
	int rn = getRuleN(currentRule);
	int rw = getRuleW(currentRule);

	if( c != 0 ){
		cellData *current = table + (			subLen * strLen + start			  ) * (rulesCount + 1);
		cellData *left	  = table + (				 k * strLen + start			  ) * (rulesCount + 1);
		cellData *right	  = table + ( (subLen - k - 1) * strLen + (k + start + 1) ) * (rulesCount + 1);
		int count1 = left[0];
		int count2 = right[0];

		if(count1 > 0 && count2 > 0){
			for(int m = 0; m < count1; m++){
				for(int n = 0; n < count2; n++){
					if ( getCellRuleTop(rules, left, m) == b && getCellRuleTop(rules, right, n) == c ){
						cellData cellData1 = left[ left[0] - m ];
						cellData cellData2 = right[ right[0] - n ];

						/*
						int lState1 = getDataS(cellData1);
						int lbl1 = getDataN(cellData1);
						int weight1 = getDataW(cellData1);

						int lState2 = getDataS(cellData2);
						int lbl2 = getDataN(cellData2);
						int weight2 = getDataW(cellData2);

						int newLabel, newlState, newWeight
						*/

						cellData currentElem = buildData(k,getRuleA(currentRule),0,0,0);

						current[0]++;
						current[ getRuleA(currentRule) ] = currentElem;
					}
				}
			}
		}
	}
}

__global__ void elem(rule* rules, int rulesCount, int* index, int strLen, int subLen, cellData* table){
	int nonTerminal = threadIdx.x;
	int i = index[nonTerminal] + threadIdx.y;
	if( i >= index[nonTerminal+1] ) return;
	int start = blockIdx.x;

	for(int k = 0; k < subLen; k++){
		processRule(rules, rulesCount, rules[i], start, k, strLen, subLen, table);
	}
}

__global__ void compress(int rulesCount, int strLen, int subLen, cellData* table){
	int start = blockIdx.x;

	cellData *current = table + (subLen * strLen + start) * (rulesCount + 1);

	current[0]=0;
	int j = 1;
	for(int i = 1; i <= rulesCount; i++ ){
		if(current[i]){
			current[j] = current[i];
			j++;
			//current[i] = 0;
			current[0]++;
		}
	}
}

//__global__ void fillRow(rule* rules, int rulesCount, int strLen, int subLen, cellData* table){
//	for(int start = 0; start < strLen - subLen; start++){
//		elem(rules, rulesCount, start, strLen, subLen, table);
//	}
//}

__host__ void fillTable(rule* rules, int rulesCount, int* index, int nCount, int strLen, cellData* table){
    cellData *dev_table = 0;
	rule *dev_rules = 0;
	int *dev_index = 0;

	int table_size = strLen * strLen * (rulesCount+1);

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);

    cudaStatus = hipMalloc((void**)&dev_table, table_size * sizeof(cellData));
    cudaStatus = hipMalloc((void**)&dev_rules, rulesCount * sizeof(rule));
    cudaStatus = hipMalloc((void**)&dev_index, (nCount + 1) * sizeof(int));
    cudaStatus = hipMemcpy(dev_table, table, table_size * sizeof(cellData), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_rules, rules, rulesCount * sizeof(rule), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_index, index, (nCount + 1) * sizeof(int), hipMemcpyHostToDevice);

	for(int subLen = 1; subLen <= strLen; subLen++){
		elem<<<strLen-subLen, dim3(nCount,2,1)>>>(dev_rules, rulesCount, dev_index, strLen, subLen, dev_table);
		compress<<<strLen-subLen, 1>>>(rulesCount, strLen, subLen, dev_table);
	}

    cudaStatus = hipDeviceSynchronize();
    cudaStatus = hipMemcpy(table, dev_table, table_size * sizeof(cellData), hipMemcpyDeviceToHost);

    hipFree(dev_table);
    hipFree(dev_rules);
    hipFree(dev_index);
}

int main(){
	const int rulesCount = 5;
	const int wordLen = 5;
	const int nCount = 4;

	rule *rules = new rule[rulesCount];
	rules[0] = buildRule(1,2,1,0,0);
	rules[1] = buildRule(2,3,3,0,0);
	rules[2] = buildRule(3,1,2,0,0);
	rules[3] = buildRule(4,1,2,0,0);
	rules[4] = buildRule(4,2,3,0,0);
	//rules[5] = buildRule(1,11,0,0,0);
	//rules[6] = buildRule(2,12,0,0,0);
	//rules[7] = buildRule(3,11,0,0,0);

	int index[nCount+1] = {0,1,2,3,rulesCount};

	cellData* table = new cellData[wordLen * wordLen * (rulesCount + 1)];
	for(int i = 0; i < wordLen * wordLen * (rulesCount + 1); i++)
		table[i] = 0;

	table[0*(rulesCount+1) + 0] = 1;
	table[0*(rulesCount+1) + 1] = buildData_Host(2);

	table[1*(rulesCount+1) + 0] = 2;
	table[1*(rulesCount+1) + 1] = buildData_Host(1);
	table[1*(rulesCount+1) + 2] = buildData_Host(3);

	table[2*(rulesCount+1) + 0] = 2;
	table[2*(rulesCount+1) + 1] = buildData_Host(1);
	table[2*(rulesCount+1) + 2] = buildData_Host(3);

	table[3*(rulesCount+1) + 0] = 1;
	table[3*(rulesCount+1) + 1] = buildData_Host(2);

	table[4*(rulesCount+1) + 0] = 2;
	table[4*(rulesCount+1) + 1] = buildData_Host(1);
	table[4*(rulesCount+1) + 2] = buildData_Host(3);


	fillTable(rules, rulesCount, index, nCount, wordLen, table);


	for(int i = 0; i < wordLen; i++){
		for(int j = 0; j < wordLen*(rulesCount+1); j++){
			if( !( j % (rulesCount+1) ) && j ) cout<<' ';
			if( !( j % (rulesCount+1) ) )
				cout<<(table[i*wordLen*(rulesCount+1)+j]);
			else
				cout<<(table[i*wordLen*(rulesCount+1)+j] >> 32 & 0xFFFF);		
			if( !( j % (rulesCount+1) ) ) cout<<' ';
		}
		cout<<endl;
	}

	system("pause");
	return 0;
}